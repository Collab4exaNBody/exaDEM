#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <exanb/core/xform.h>

#include <memory>

#include <exaDEM/contact_force_parameters.h>
#include <exaDEM/compute_contact_force.h>
#include <exaDEM/interaction/interaction.hpp>
#include <exaDEM/classifier/interactionSOA.hpp>
#include <exaDEM/classifier/interactionAOS.hpp>
#include <exaDEM/interaction/grid_cell_interaction.hpp>
#include <exaDEM/classifier/classifier.hpp>
#include <exaDEM/classifier/classifier_for_all.hpp>
#include <exaDEM/itools/itools.hpp>
#include <exaDEM/shapes.hpp>
#include <exaDEM/shape_detection.hpp>
#include <exaDEM/shape_detection_driver.hpp>
#include <exaDEM/mutexes.h>
#include <exaDEM/drivers.h>
#include <exaDEM/contact_sphere.h>
#include <exaDEM/multimat_cp.h>

namespace exaDEM
{
  using namespace exanb;
  using namespace sphere;

  template <bool multimat, bool cohesive, typename GridT, class = AssertGridHasFields<GridT, field::_vx, field::_vy, field::_vz, field::_mom, field::_orient, field::_vrot, field::_radius>> 
    class ComputeContactClassifierSphere : public OperatorNode
  {
    // attributes processed during computation
    using ComputeFields = FieldSet<field::_vrot, field::_arot>;
    static constexpr ComputeFields compute_field_set{};

    ADD_SLOT(GridT, grid, INPUT_OUTPUT, REQUIRED);
    ADD_SLOT(Domain , domain, INPUT , REQUIRED );
    ADD_SLOT(ContactParams, config, INPUT, OPTIONAL, DocString{"Contact parameters for sphere interactions"});      // can be re-used for to dump contact network
    ADD_SLOT(ContactParams, config_driver, INPUT, OPTIONAL, DocString{"Contact parameters for drivers, optional"}); // can be re-used for to dump contact network
    ADD_SLOT(ContactParamsMultiMat<ContactParams>, multimat_cp, INPUT, DocString{"List of contact parameters for simulations with multiple materials"});
    ADD_SLOT(double, dt, INPUT, REQUIRED, DocString{"Time step value"});
    ADD_SLOT(bool, symetric, INPUT_OUTPUT, REQUIRED, DocString{"Activate the use of symetric feature (contact law)"});
    ADD_SLOT(Drivers, drivers, INPUT, REQUIRED, DocString{"List of Drivers {Cylinder, Surface, Ball, Mesh}"});
    ADD_SLOT(Classifier<InteractionSOA>, ic, INPUT_OUTPUT, DocString{"Interaction lists classified according to their types"});
    // analysis
    ADD_SLOT(long, timestep, INPUT, REQUIRED);
    ADD_SLOT(long, analysis_interaction_dump_frequency, INPUT, REQUIRED, DocString{"Write an interaction dump file"});
    ADD_SLOT(std::string, dir_name, INPUT, REQUIRED, DocString{"Output directory name."});
    ADD_SLOT(std::string, interaction_basename, INPUT, REQUIRED, DocString{"Write an Output file containing interactions."});
    // private
    ADD_SLOT(bool, print_warning, PRIVATE, true, DocString{"This variable is used to ensure that warning messages are displayed only once."});
    // output
    ADD_SLOT(double, max_kn, INPUT_OUTPUT, 0, DocString{"Get the highest value of the input contact force parameters kn (used for dt_critical)"});


    public:
    inline std::string documentation() const override final { return R"EOF(This operator computes forces between particles and particles/drivers using the contact law.)EOF"; }

    template<int start, int end, template<int, bool, typename> typename FuncT, typename XFormT, typename T, typename... Args>
      void loop_contact_force(Classifier<T>& classifier, XFormT& cp_xform, Args &&... args)
      {
        FuncT<start, cohesive, XFormT> contact_law;
        contact_law.xform = cp_xform;
        run_contact_law(parallel_execution_context(), start, classifier, contact_law, args...);
        if constexpr( start + 1 <= end )
        {
          loop_contact_force<start+1, end, FuncT, XFormT>(classifier, cp_xform, std::forward<Args>(args)...);
        }
      }

    template<bool is_sym, typename XFormT>
      void core(XFormT& xform)
      {
        const DriversGPUAccessor drvs = *drivers;
        auto *cells = grid->cells();

        const double time = *dt;
        auto &classifier = *ic;

        contact_law<is_sym, cohesive, XFormT> sph = {xform};
        contact_law_driver<cohesive, Cylinder, XFormT> cyl = {xform};
        contact_law_driver<cohesive, Surface, XFormT> surf = {xform};
        contact_law_driver<cohesive, Ball, XFormT> ball = {xform};

        if( !multimat ) /** single mat */
        {
          const ContactParams hkp = *config;
          ContactParams hkp_drvs{};

          if (drivers->get_size() > 0 )
          {
            hkp_drvs = *config_driver;
          }

          const SingleMatContactParamsTAccessor<ContactParams> cp = {hkp};
          const SingleMatContactParamsTAccessor<ContactParams> cp_drvs = {hkp_drvs};

          run_contact_law(parallel_execution_context(), 0, classifier, sph, cells, cp, time);
          run_contact_law(parallel_execution_context(), 4, classifier, cyl, cells, drvs, cp_drvs, time);
          run_contact_law(parallel_execution_context(), 5, classifier, surf, cells, drvs, cp_drvs, time);
          run_contact_law(parallel_execution_context(), 6, classifier, ball, cells, drvs, cp_drvs, time);

          constexpr int stl_type_start = 7;
          constexpr int stl_type_end = 9;
          loop_contact_force <stl_type_start,  stl_type_end, contact_law_stl, XFormT>(classifier, xform, cells, drvs, cp_drvs, time);
        }
        else
        {
          const auto& contact_parameters = *multimat_cp;
          const MultiMatContactParamsTAccessor<ContactParams> cp = contact_parameters.get_multimat_accessor();
          const MultiMatContactParamsTAccessor<ContactParams> cp_drvs = contact_parameters.get_drivers_accessor();
          run_contact_law(parallel_execution_context(), 0, classifier, sph, cells, cp, time);
          run_contact_law(parallel_execution_context(), 4, classifier, cyl, cells, drvs, cp_drvs, time);
          run_contact_law(parallel_execution_context(), 5, classifier, surf, cells, drvs, cp_drvs, time);
          run_contact_law(parallel_execution_context(), 6, classifier, ball, cells, drvs, cp_drvs, time);

          constexpr int stl_type_start = 7;
          constexpr int stl_type_end = 9;
          loop_contact_force <stl_type_start,  stl_type_end, contact_law_stl, XFormT>(classifier, xform, cells, drvs, cp_drvs, time);
        }
      }

    void save_results()
    {
      /** Analysis */
      const long frequency_interaction = *analysis_interaction_dump_frequency;
      bool write_interactions = (frequency_interaction > 0 && (*timestep) % frequency_interaction == 0);
      if(write_interactions)
      {
        auto &classifier = *ic;
        auto stream = itools::create_buffer(*grid, classifier);
        std::string ts = std::to_string(*timestep);
        itools::write_file(stream, *dir_name, (*interaction_basename) + ts);
      }
    }

    void check_slots()
    {
      bool pw = true;

      /** polyhedron interactions are defined while the contact sphere operator is used */
      {
        auto &classifier = *ic;
        for(int i = 1 ; i <= 3 ; i++)
        {
          auto& interactions = classifier.get_wave(i);
          if(interactions.size() > 0)
          {
            lout << "[ERROR]: the contact operator for spheres is being used, but polyhedron interactions are defined." << std::endl;
            lout << "         Please, use contact_polyhedron operators. " << std::endl;    
            std::exit(0);
          }
        }
      }

      /** Some check mutlimat versus singlemat */
      if constexpr  (multimat) /** Multiple materials */
      {
        if( !multimat_cp.has_value() )
        {
          lout << "\033[1;31m[ERROR]: You are using the multi-material contact force model, but the contact law parameters have not been defined. "
            << "Please specify the parameter values for each material pair using the operator \"multimat_contact_params\".\033[0m"
            << std::endl;
          std::exit(EXIT_FAILURE);
        }
        if( *print_warning && config.has_value() ) 
        {
          lout << "\033[1;33m[WARNING]: You are using the multi-material contact force operator, but you have also defined the input slot \"config\" which is intended for the single-material version. This slot will be ignored.\033[0m"
            << std::endl;
          pw = false;
        }
        if( *print_warning && config_driver.has_value() ) 
        {
          lout << "\033[1;33m[WARNING]: You are using the multi-material contact force operator, but you have also defined the input slot \"config_driver\" which is intended for the single-material version. This slot will be ignored.\033[0m"
            << std::endl;
          pw = false;
        }
      }

      if constexpr  (!multimat) /** Single material */
      {
        if( !config.has_value() ) 
        {
          lout << "\033[1;31m[ERROR]: The input slot \"config\" is not defined, yet the single-material version of the contact operator is being used. "
            << "Please specify the \"config\" input slot, and use the \"config_driver\" slot if you want to define a contact law between a particle and a driver.\033[0m"
            << std::endl;
        }
        if( multimat_cp.has_value() )
        {
          lout << "\033[1;33m[WARNING] You have defined a list of contact law parameters for different material types, "
            << "but you are using the version that only considers the parameter defined in the \"config\" input slot. "
            << "The parameter list will be ignored. If you want to use it, please use the operator "
            << "\"contact_sphere_multimat\" or \"contact_sphere_multimat_with_cohesion\".\033[0m"
            << std::endl;
          pw = false;
        }
        /** Some global checks */
        /** Is cohesive force define while it's not used */
        if constexpr (!cohesive)
        {
          if(config->dncut > 0)
          {
            lout << "[ERROR]: dncut is != 0 while the cohesive force is not used." << std::endl;
            lout << "         Please, use contact_sphere_with_cohesion operator." << std::endl;
            std::exit(0);
          }
          if(drivers->get_size() > 0 && config_driver->dncut > 0)
          {
            lout << "[ERROR]: dncut is != 0 while the cohesive force is not used." << std::endl;
            lout << "         Please, use contact_sphere_with_cohesion operator." << std::endl;
            std::exit(0);
          }
        }
      }
      *print_warning = pw;
    }

    /** fill highest kn */
    void scan_kn()
    {
      if(!(*print_warning)) return;
      double kn = 0.0;
      if(config.has_value()) kn = std::max(kn, config->kn);
      if(config_driver.has_value()) kn = std::max(kn, config->kn);
      if(multimat_cp.has_value())
      {
        auto get_max_kn = [&kn] (const ContactParams& cp) -> void { kn = std::max(kn, cp.kn); };
        multimat_cp->apply(get_max_kn);
      }
      *max_kn = kn;
    }

    inline void execute() override final
    {
      check_slots();
      scan_kn();

      if (grid->number_of_cells() == 0)
      {
        return;
      }

      if(!domain->xform_is_identity())
      {
        LinearXForm cp_xform = {domain->xform()};
        if(*symetric) core<true>(cp_xform);
        else core<false>(cp_xform);
      }
      else     
      {
        NullXForm cp_xform;
        if(*symetric) core<true>(cp_xform);
        else core<false>(cp_xform);
      }     

      save_results();
    }
  };

  template <class GridT> using ComputeContactSphereSingleMatTmpl = ComputeContactClassifierSphere<false, false, GridT>;
  template <class GridT> using ComputeContactSphereSingleMatCohesiveTmpl = ComputeContactClassifierSphere<false, true, GridT>;
  template <class GridT> using ComputeContactSphereMultiMatTmpl = ComputeContactClassifierSphere<true, false, GridT>;
  template <class GridT> using ComputeContactSphereMultiMatCohesiveTmpl = ComputeContactClassifierSphere<true, true, GridT>;

  // === register factories ===
  ONIKA_AUTORUN_INIT(contact_force_sphere) { OperatorNodeFactory::instance()->register_factory("contact_sphere", make_grid_variant_operator<ComputeContactSphereSingleMatTmpl>); }
  ONIKA_AUTORUN_INIT(contact_force_sphere_sm) { OperatorNodeFactory::instance()->register_factory("contact_sphere_singlemat", make_grid_variant_operator<ComputeContactSphereSingleMatTmpl>); }
  ONIKA_AUTORUN_INIT(contact_force_sphere_with_cohesion) { OperatorNodeFactory::instance()->register_factory("contact_sphere_with_cohesion", make_grid_variant_operator<ComputeContactSphereSingleMatCohesiveTmpl>); }
  ONIKA_AUTORUN_INIT(contact_force_sphere_sm_with_cohesion) { OperatorNodeFactory::instance()->register_factory("contact_sphere_singlemat_with_cohesion", make_grid_variant_operator<ComputeContactSphereSingleMatCohesiveTmpl>); }
  ONIKA_AUTORUN_INIT(contact_force_sphere_mm) { OperatorNodeFactory::instance()->register_factory("contact_sphere_multimat", make_grid_variant_operator<ComputeContactSphereMultiMatTmpl>); }
  ONIKA_AUTORUN_INIT(contact_force_sphere_mm_with_cohesion) { OperatorNodeFactory::instance()->register_factory("contact_sphere_multimat_with_cohesion", make_grid_variant_operator<ComputeContactSphereMultiMatCohesiveTmpl>); }
} // namespace exaDEM
