#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements.  See the NOTICE file
distributed with this work for additional information
regarding copyright ownership.  The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License.  You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/grid.h>
#include <exanb/compute/compute_cell_particles.h>
#include <memory>
#include <exaDEM/set_fields.h>

namespace exaDEM
{
  using namespace exanb;

  template <typename GridT, class = AssertGridHasFields<GridT, field::_radius>> class SetRadius : public OperatorNode
  {
    static constexpr double default_radius = 0.5;
    using ComputeFields = FieldSet<field::_radius>;
    using ComputeRegionFields = FieldSet<field::_rx, field::_ry, field::_rz, field::_id, field::_radius>;
    static constexpr ComputeFields compute_field_set{};
    static constexpr ComputeRegionFields compute_region_field_set{};

    ADD_SLOT(GridT, grid, INPUT_OUTPUT);
    ADD_SLOT(double, rad, INPUT, default_radius, DocString{"default radius value for all particles"});
    ADD_SLOT(ParticleRegions, particle_regions, INPUT, OPTIONAL);
    ADD_SLOT(ParticleRegionCSG, region, INPUT, OPTIONAL);
    ADD_SLOT(double, rcut_max, INPUT_OUTPUT, DocString{"rcut_max"});

  public:
    inline std::string documentation() const override final
    {
      return R"EOF(
        This operator sets the radius value for every particles.
        )EOF";
    }

    inline void execute() override final
    {
      if (rcut_max.has_value())
      {
        *rcut_max = std::max(*rcut_max, 2 * (*rad));
      }
      else
      {
        *rcut_max = 2 * (*rad);
      }

      if (region.has_value())
      {
        if (!particle_regions.has_value())
        {
          fatal_error() << "Region is defined, but particle_regions has no value" << std::endl;
        }

        if (region->m_nb_operands == 0)
        {
          ldbg << "rebuild CSG from expr " << region->m_user_expr << std::endl;
          region->build_from_expression_string(particle_regions->data(), particle_regions->size());
        }

        ParticleRegionCSGShallowCopy prcsg = *region;
        SetRegionFunctor<double> func = {prcsg, {*rad}};
        compute_cell_particles(*grid, false, func, compute_region_field_set, parallel_execution_context());
      }
      else
      {
        SetFunctor<double> func = {{*rad}};
        compute_cell_particles(*grid, false, func, compute_field_set, parallel_execution_context());
      }
    }
  };

  template <class GridT> using SetRadiusTmpl = SetRadius<GridT>;

  // === register factories ===
  CONSTRUCTOR_FUNCTION { OperatorNodeFactory::instance()->register_factory("set_radius", make_grid_variant_operator<SetRadiusTmpl>); }

} // namespace exaDEM
