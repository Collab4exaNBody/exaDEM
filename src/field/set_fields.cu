#include "hip/hip_runtime.h"
/*
   Licensed to the Apache Software Foundation (ASF) under one
   or more contributor license agreements.  See the NOTICE file
   distributed with this work for additional information
   regarding copyright ownership.  The ASF licenses this file
   to you under the Apache License, Version 2.0 (the
   "License"); you may not use this file except in compliance
   with the License.  You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
 */

#include <onika/scg/operator.h>
#include <onika/scg/operator_slot.h>
#include <onika/scg/operator_factory.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/core/parallel_grid_algorithm.h>
#include <exanb/core/grid.h>
#include <exanb/core/particle_type_id.h>
#include <memory>
#include <exaDEM/shapes.hpp>
#include <exaDEM/set_fields.h>
#include <exaDEM/random_quaternion.h>

namespace exaDEM
{

  struct jammy
  {
    jammy(double var) { dist = std::normal_distribution<>(0, var); }

    inline int operator()(double &val)
    {
      val += dist(seed);
      seed();
      return 0;
    }

    inline int operator()(Vec3d &val)
    {
      val.x += dist(seed);
      seed();
      val.y += dist(seed);
      seed();
      val.z += dist(seed);
      seed();
      return 0;
    }

    std::normal_distribution<> dist;
    std::default_random_engine seed;
  };


  struct field_manager
  {
    bool set_t = false; // type
    bool set_d = false; // density
    bool set_v = false; // velocity
    bool set_rnd_v = false;
    bool set_r = false; // radius
    bool set_q = false; // quaternion
    bool set_rnd_q = false;
    bool set_i = true; // inertia (should be to true)
    bool set_ang_v = false;
    bool set_rnd_ang_v = false;
  };

  using namespace exanb;

  template <typename GridT, class = AssertGridHasFields<GridT, field::_type>> class SetFields : public OperatorNode
  {
    // fields : vx, vy, vz, mass, radius, anv, inertia, quat
    using ComputeFields = FieldSet<field::_type, field::_vx, field::_vy, field::_vz, field::_mass, field::_radius, field::_vrot, field::_inertia, field::_orient>;
    using ComputeRegionFields = FieldSet<field::_rx, field::_ry, field::_rz, field::_id, field::_type, field::_vx, field::_vy, field::_vz, field::_mass, field::_radius, field::_vrot, field::_inertia, field::_orient>;
    static constexpr ComputeFields compute_fields{};
    static constexpr ComputeRegionFields compute_region_fields{};

    ADD_SLOT(GridT, grid, INPUT_OUTPUT);
    // vector version
    ADD_SLOT(std::vector<double>, density, INPUT, OPTIONAL, DocString{"List of density values. If not defined, density is 1"});
    ADD_SLOT(std::vector<double>, radius, INPUT, OPTIONAL, DocString{"List of radius values. If not defined, radius is 0.5 for spheres, do not define it for polyhedra."});
    ADD_SLOT(std::vector<Vec3d>, velocity, INPUT, OPTIONAL, DocString{"List of velocity values. If not defined, velocity is [0,0,0]."});
    ADD_SLOT(std::vector<double>, sigma_velocity, INPUT, OPTIONAL, DocString{"Standard deviation (sigma). If not defined, the normal distribution is not applied."});
    ADD_SLOT(std::vector<Vec3d>, angular_velocity, INPUT, OPTIONAL, DocString{"List of angular velocity values. If not defined, angular velocity is [0,0,0]."});
    ADD_SLOT(std::vector<double>, sigma_angular_velocity, INPUT, OPTIONAL, DocString{"Standard deviation (sigma). If not defined, the normal distribution is not applied."});
    ADD_SLOT(std::vector<Quaternion>, quaternion, INPUT, OPTIONAL, DocString{"List of orientations. If not defined, quaternion is [w = 1,0,0,0]"});
    ADD_SLOT(std::vector<bool>, random_quaternion, INPUT, OPTIONAL, DocString{"Choice if the orientation is random or not. If not defined, random is false."});
    ADD_SLOT(ParticleTypeMap, particle_type_map, INPUT, REQUIRED );
    ADD_SLOT(std::vector<std::string>, type, INPUT, REQUIRED, DocString{"Particle type names"});

    // outputs
    ADD_SLOT(double, rcut_max, INPUT_OUTPUT, DocString{"rcut_max"});

    // others
    ADD_SLOT(bool, polyhedra, INPUT, REQUIRED, DocString{"Define if the kind of particles is polyhedron or sphere."});
    ADD_SLOT(ParticleRegions, particle_regions, INPUT, OPTIONAL);
    ADD_SLOT(ParticleRegionCSG, region, INPUT, OPTIONAL);
    ADD_SLOT(shapes, shapes_collection, INPUT, OPTIONAL, DocString{"Collection of shapes"});

    // -----------------------------------------------
    // ----------- Operator documentation ------------
    inline std::string documentation() const override final
    {
      return R"EOF(
        This operator fills type id to all particles. 
        )EOF";
    }

    void check_slots()
    {
      if(grid->number_of_cells() == 0)
      {
        lout << "\033[1;31m[set_fields, ERROR] the grid is not defined. Please define a grid before calling set_fields.\033[0m" << std::endl;
        std::exit(EXIT_FAILURE);
      }

      if(shapes_collection.has_value())
      {
        if(!(*polyhedra))
        {
          lout << "[set_fields, ERROR] Shapes are defined in sphere mode" << std::endl;
          std::exit(EXIT_FAILURE);  
        }
        size_t size_shps = shapes_collection->get_size();
        if(size_shps == 0 && (*polyhedra))
        {
          lout << "[set_fields, ERROR] You are defining polyhedra without using shapes" << std::endl;
          std::exit(EXIT_FAILURE);  
        }
      }
    }

    public:
    inline void execute() override final
    {

      check_slots();

      const auto& type_map = *particle_type_map; 
      const auto& types = *type;

      bool is_region  = region.has_value();

      field_manager mat; // multi-materials
      mat.set_t         = type.has_value();
      mat.set_d         = density.has_value();
      mat.set_r         = radius.has_value();
      mat.set_v         = velocity.has_value();
      mat.set_rnd_v     = sigma_velocity.has_value();
      mat.set_ang_v     = angular_velocity.has_value();
      mat.set_rnd_ang_v = sigma_angular_velocity.has_value();
      mat.set_q         = quaternion.has_value();
      mat.set_rnd_q     = random_quaternion.has_value();


      lout << "======= Particle Fields =========" << std::endl;
      for(size_t i = 0 ; i < types.size(); i++)
      {
        std::string type_name = types[i];
        if( type_map.find(type_name) == type_map.end())
        {
          lout << "The type [" << type_name << "] is not defined" << std::endl;
          lout << "Available types are = ";
          for(auto& it : type_map) lout << it.first << " ";
          lout << std::endl;
          std::exit(EXIT_FAILURE);  
        }
        int64_t type_id = type_map.at(type_name);
        // default values;
        double vx = 0;
        double vy = 0;
        double vz = 0;
        double r = 1.0; // it will be replaced if the polyhedra is to true.
        double d = 1.0;
        double m = 1.0;
        Vec3d ang_v = {0,0,0};
        Quaternion quat = {1,0,0,0};
        Vec3d inertia;
        double sigma_v, sigma_ang_v;

        if(mat.set_d) { auto& dd = *density; d = dd[i]; }
        if(mat.set_v) { auto& vv = *velocity; const Vec3d& v = vv[i]; vx = v.x; vy = v.y; vz = v.z; }
        if(mat.set_ang_v) { auto& ang_vv = *angular_velocity; ang_v = ang_vv[i]; }
        if(mat.set_q) { auto& qq = *quaternion; quat = qq[i]; }


        lout << "[>> "<<type_name<<" <<]" << std::endl;;
        lout << "Velocity         = (" << vx << "," << vy << "," << vz << ") ";
        if(mat.set_rnd_v)
        {
          sigma_v = (*sigma_velocity)[i];
          lout << ", standart deviation (sigma): " << sigma_v;
        }
        lout << std::endl; 
        lout << "Angular velocity = " << ang_v;
        if(mat.set_rnd_ang_v)
        {
          sigma_ang_v = (*sigma_angular_velocity)[i];
          lout << ", standart deviation (sigma): " << sigma_ang_v;
        }
        lout << std::endl; 
        lout << "Density          = " << d << std::endl;;
        if( !mat.set_rnd_q ) lout << "Quaternion       = [w: " << quat.w << ", v: (" << quat.x << "," << quat.y << "," << quat.z << ")]" ;
        else lout << "Quaternion       = random";
        lout << std::endl; 


        if(*polyhedra)
        {
          const shapes& shps = *shapes_collection;
          const auto& shp = shps[type_id];
          if( type_id >= shps.get_size() || shp->m_name != type_name ) {
             
             lout << "[set_fields, ERROR]  We can't find the shape related to the type "  <<type_name << ". Please verify that you have load all shape files." << std::endl; 
          }
          m         = d * shp->get_volume();
          inertia   = m * shp->get_Im();

          if( mat.set_r ) { lout << "[set_fields, WARNING] The radius slot is ignored when using polyhedra, it is automaticly deducted from the shape file."<< std::endl; }
          r = shp->compute_max_rcut();
          *rcut_max = std::max(*rcut_max, 2 * r); // r * maxrcut
          lout << "Radius (poly)    = " << r << std::endl;;
          lout << "Mass             = " << m << std::endl;
          lout << "Inertia          = " << inertia << std::endl;
        }
        else // spheres
        {
          if(!mat.set_r) { lout << "[set_fields, ERROR] You should define a radius: radius: \"[1.0]\"" ; std::exit(EXIT_FAILURE); }
          else
          { 
            auto& rr = *radius; 
            r = rr[i]; 
          }
          *rcut_max = std::max(*rcut_max, 2 * r); // r * maxrcut
          const double pi = 4 * std::atan(1);
          const double V = ((4.0)/(3.0)) * pi * r * r * r;
          m = V  * d ;
          const double inertia_value = 0.4 * m * r * r;
          inertia = {inertia_value, inertia_value, inertia_value};
          lout << "Radius           = " << r << std::endl;
          lout << "Mass             = " << m << std::endl;
          lout << "Inertia          = " << inertia << std::endl;
        }

        if (is_region)
        {
          ParticleRegionCSGShallowCopy prcsg = *region;
          if (!particle_regions.has_value())
          {
            fatal_error() << "Region is defined, but particle_regions has no value" << std::endl;
          }

          if (region->m_nb_operands == 0)
          {
            ldbg << "rebuild CSG from expr " << region->m_user_expr << std::endl;
            region->build_from_expression_string(particle_regions->data(), particle_regions->size());
          }

          // fields : vx, vy, vz, mass, radius, anv, inertia, quat
          FilteredSetRegionFunctor<double,double, double, double, double, Vec3d, Vec3d, Quaternion> func = {prcsg, uint32_t(type_id), {vx, vy, vz, m, r, ang_v, inertia, quat}};
          compute_cell_particles(*grid, false, func, compute_region_fields, parallel_execution_context());

          if(mat.set_rnd_v)
          {
            jammy gen(sigma_v);
            FieldSet<field::_rx, field::_ry, field::_rz, field::_vx, field::_vy, field::_vz> compute_rnd_v;
            GenSetRegionFunctor<jammy> generator = {prcsg, gen};
            compute_cell_particles(*grid, false, generator, compute_rnd_v, parallel_execution_context());
          }

          if(mat.set_rnd_ang_v)
          {
            jammy gen(sigma_ang_v);
            FieldSet<field::_rx, field::_ry, field::_rz, field::_vrot> compute_rnd_ang_v;
            GenSetRegionFunctor<jammy> generator = {prcsg, gen};
            compute_cell_particles(*grid, false, generator, compute_rnd_ang_v, parallel_execution_context());
          }

          if(mat.set_rnd_q) /** Random Quaternion */
          {
            FieldSet<field::_rx, field::_ry, field::_rz, field::_id, field::_orient> compute_orient;
            RandomQuaternionFunctor RndQuatFunc = {prcsg};
            compute_cell_particles(*grid, false, RndQuatFunc, compute_orient, parallel_execution_context());
          }
        }
        else // no region
        {
          FilteredSetFunctor<double, double, double, double, double, Vec3d, Vec3d, Quaternion> func = {uint32_t(type_id), {vx, vy, vz, m, r, ang_v, inertia, quat}};
          compute_cell_particles(*grid, false, func, compute_fields, parallel_execution_context());
          if(mat.set_rnd_v)
          {
            jammy gen(sigma_v);
            FieldSet<field::_vx, field::_vy, field::_vz> compute_rnd_v;
            GenSetFunctor<jammy> generator = {gen};
            compute_cell_particles(*grid, false, generator, compute_rnd_v, parallel_execution_context());
          }

          if(mat.set_rnd_q) /** Random Quaternion */
          {
            FieldSet<field::_orient> compute_orient;
            RandomQuaternionFunctor RndQuatFunc = {};
            compute_cell_particles(*grid, false, RndQuatFunc, compute_orient, parallel_execution_context());
          }
        }
      }
      lout << "=================================" << std::endl;
    }
  };

  template <class GridT> using SetFieldsTmpl = SetFields<GridT>;

  // === register factories ===
  ONIKA_AUTORUN_INIT(set_fields) { OperatorNodeFactory::instance()->register_factory("set_fields", make_grid_variant_operator<SetFieldsTmpl>); }

} // namespace exaDEM
